#include "raft_radon_gpu.h"
#include "raft_radon_gpu_function.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define INIC -1.0

/*
Autor: Joao Carlos Cerqueira	email: jc.cerqueira13@gmail.com
*/

extern "C" {
void raft_radon_slantstack_gpu(float* h_output, float* h_input, int sizeImage, int nrays, int nangles)
{
	float *d_output, *d_input;

	// Allocate GPU buffers for the output sinogram
	hipMalloc(&d_output, sizeof(float) * nrays * nangles);
	
	// Allocate GPU memory for input image and copy
	hipMalloc(&d_input, sizeof(float) * sizeImage * sizeImage);
	hipMemcpy(d_input, h_input, sizeof(float) * sizeImage * sizeImage, hipMemcpyHostToDevice);	

	raft_radon_gpu_function(d_output, d_input, sizeImage, nrays, nangles, 0.0);

	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(h_output, d_output, sizeof(float) * nrays * nangles, hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);
	hipDeviceReset();
}
}
