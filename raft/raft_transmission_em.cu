#include "raft_cuda_aux.h"
#include "raft_backprojection_gpu_function.h"
#include "raft_radon_gpu_function.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/*
Autor: Joao Carlos Cerqueira	email: jc.cerqueira13@gmail.com

Implementacao de um algoritmo iterativo de reconstrucao restrito
a problemas de tomografia baseados em transmissao.
*/

extern "C"{
void raft_tr_em_gpu(float *output, float *rawsino, float *flat, int sizeImage, int nrays, int nangles, int niter)
{
	int i;	

	int SinoMem = sizeof(float)*nrays*nangles;
	int ImageMem = sizeof(float)*sizeImage*sizeImage;	

	float *d_image1, *d_image2, *d_image3, *d_flat, *d_sino1, *d_sino2, *d_sino3, *d_Sino;
	float *img_inicial;

// CUDA LINEAR MEMORY
	hipMalloc(&d_sino1, SinoMem);
	hipMalloc(&d_sino2, SinoMem);
	hipMalloc(&d_sino3, SinoMem);	
	hipMalloc(&d_Sino, SinoMem);	
	hipMalloc(&d_flat, SinoMem);
	hipMalloc(&d_image1, ImageMem);
	hipMalloc(&d_image2, ImageMem);
	hipMalloc(&d_image3, ImageMem);

// IMAGEM INICIAL
/* Futuramente, pode ser interessante ter a imagem inicial como uma variável */
	img_inicial = (float *)malloc(ImageMem);
	for (i = 0; i < (sizeImage*sizeImage); i++)		
		img_inicial[i] = 1.0;

	hipMemcpy(d_image1, img_inicial, ImageMem, hipMemcpyHostToDevice);


// COPIAS DA CPU PARA A GPU
	hipMemcpy(d_flat, flat, SinoMem, hipMemcpyHostToDevice);
	hipMemcpy(d_Sino, rawsino, SinoMem, hipMemcpyHostToDevice);


	raft_backprojection_gpu_function(d_image3, d_Sino, sizeImage, nrays, nangles);

	for(i=0; i<niter; i++){
		raft_radon_gpu_function(d_sino1, d_image1, sizeImage, nrays, nangles, -1.0);
		mtx_elementwise_mult(d_sino1, d_sino1, d_flat, nrays*nangles);
		raft_backprojection_gpu_function(d_image2, d_sino1, sizeImage, nrays, nangles);
		mtx_elementwise_div(d_image2, d_image2, d_image3, sizeImage*sizeImage);
		mtx_elementwise_mult(d_image1, d_image1, d_image2, sizeImage*sizeImage);

		mtx_elementwise_mult(d_image1, d_image1, d_image2, sizeImage*sizeImage);
	
	}

// COPIA DO RESULTADO DA GPU PARA A CPU
	hipMemcpy(output , d_image1 , ImageMem , hipMemcpyDeviceToHost);

	hipFree(d_sino1);
	hipFree(d_sino2);
	hipFree(d_sino3);
	hipFree(d_Sino);
	hipFree(d_flat);
	hipFree(d_image1);
	hipFree(d_image2);
	hipFree(d_image3);
	free(img_inicial);

	return;
}// END VOID
}
