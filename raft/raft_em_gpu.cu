#include "raft_cuda_aux.h"
#include "raft_backprojection_gpu_function.h"
#include "raft_radon_gpu_function.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define THREADS_BLOCK_EM 256

/*Autor: Joao Carlos Cerqueira	email: jc.cerqueira13@gmail.com

A funcao 'void raft_em_gpu' implementa o metodo de reconstrucao iterativo
EM, onde 'niter' eh o numero de iteracoes. Este metodo de reconstrucao é
limitado a tomografias baseadas em emissao. Para inversoes de tomografias
baseadas em transmissao, utilize a funcao 'void raft_tr_em_gpu'
*/


extern "C" {
void raft_em_gpu(float *output, float *sino, int sizeImage, int nrays, int nangles, int niter){
	int i, nit;	
	
	int SinoMem = sizeof(float)*nrays*nangles;
	int ImageMem = sizeof(float)*sizeImage*sizeImage;	

	float *d_image1, *d_image2, *d_image3, *d_sino1, *d_sino2, *d_Sino;
	float *h_sino_ones, *d_bp_ones;
	float *img_inicial;
	
	
	//	SET CUDA LINEAR MEMORY
	hipMalloc(&d_sino1, SinoMem);
	hipMalloc(&d_sino2, SinoMem);	
	hipMalloc(&d_Sino, SinoMem);	
	hipMalloc(&d_image1, ImageMem);
	hipMalloc(&d_image2, ImageMem);
	hipMalloc(&d_image3, ImageMem);
	hipMalloc(&d_bp_ones, ImageMem);

	
	//	Imagem inicial (provisório)
	img_inicial	= (float *)malloc(ImageMem);
	for (i = 0; i < (sizeImage*sizeImage); i++)		
		img_inicial[i] = 1.0;

	hipMemcpy(d_image1, img_inicial, ImageMem, hipMemcpyHostToDevice);

	// Sinogram of ones
	h_sino_ones = (float *)malloc(SinoMem);
	for(i = 0; i < (nrays*nangles); i++)
		h_sino_ones[i] = 1;

	hipMemcpy(d_sino1, h_sino_ones, SinoMem, hipMemcpyHostToDevice);
	


	//	BP OF ONES and INPUT SINOGRAM COPY
	raft_backprojection_gpu_function(d_bp_ones, d_sino1, sizeImage, nrays, nangles);

	hipMemcpy(d_Sino, sino, SinoMem, hipMemcpyHostToDevice);	

	//	THE FOR LOOP
	for(nit = 0; nit < niter; nit++){
		raft_radon_gpu_function(d_sino1, d_image1, sizeImage, nrays, nangles, 0.0);
		mtx_elementwise_div(d_sino2, d_Sino, d_sino1, nrays*nangles);
		raft_backprojection_gpu_function(d_image2, d_sino2, sizeImage, nrays, nangles);
		mtx_elementwise_div(d_image3, d_image2, d_bp_ones, sizeImage*sizeImage);
		mtx_elementwise_mult(d_image1, d_image3, d_image1, sizeImage*sizeImage);
	}

	hipMemcpy(output , d_image1 , ImageMem , hipMemcpyDeviceToHost);

	
	hipFree(d_image1);
	hipFree(d_image2);
	hipFree(d_image3);
	hipFree(d_sino1);
	hipFree(d_sino2);
	hipFree(d_Sino);
	hipFree(d_bp_ones);

	
	free(img_inicial);
	free(h_sino_ones);

	return;
}
}
