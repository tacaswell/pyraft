#include "hip/hip_runtime.h"
#include "raft_backprojection_gpu_function.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define PI 3.141592653589793238462643383279502884
#define INIC -1.0
/* The TPBXb and TPBYb are parameters that can be changed and may interfere the performance */
#define TPBXb 32
#define TPBYb 32

texture<float, hipTextureType2D, hipReadModeElementType> texSino;


extern "C" {
__global__ void raft_backprojection_gpu_kernel(float *image, int wdI, int nrays, int nangles, float delta, float dt, float dth)
{
  int i, j, T;
  float t, cs1, cs2, cs3, cs4, k;
  float	x, y;
  float cosk, sink;

  i = 2*(blockDim.x * blockIdx.x + threadIdx.x);
  j = 2*(blockDim.y * blockIdx.y + threadIdx.y);

  if ( ((i+1)<wdI) && ((j+1) < wdI) ){

	  cs1 = 0;
	  cs2 = 0;
	  cs3 = 0;
	  cs4 = 0;

	  for(k=0; k < (nangles); k++)
	  {
		  sincosf(k * dth, &sink, &cosk);
		  ///////////////////////////
		  x = (float)INIC + i * delta;
		  y = (float)INIC + j * delta;
	      t = x*cosk + y*sink;
	      T = (float)((t + 1)/dt);
	      //if(T >= 0 && T <= (nrays-1))
		  	  cs1 = cs1 + tex2D(texSino, k + 0.5f, T + 0.5f);
		  //////////////////////////
		  x = (float)INIC + (i+1) * delta;
		  y = (float)INIC + j * delta;
	      t = x*cosk + y*sink;
	      T = (float)((t + 1)/dt);
	      //if(T >= 0 && T <= (nrays-1))
		  	  cs2 = cs2 + tex2D(texSino, k + 0.5f, T + 0.5f);
		  //////////////////////////
		  x = (float)INIC + i * delta;
		  y = (float)INIC + (j+1) * delta;
	      t = x*cosk + y*sink;
	      T = (float)((t + 1)/dt);
	      //if(T >= 0 && T <= (nrays-1))
		  	  cs3 = cs3 + tex2D(texSino, k + 0.5f, T + 0.5f);
		  //////////////////////////
		  x = (float)INIC + (i+1) * delta;
		  y = (float)INIC + (j+1) * delta;
	      t = x*cosk + y*sink;
	      T = (float)((t + 1)/dt);
	      //if(T >= 0 && T <= (nrays-1))
		  	  cs4 = cs4 + tex2D(texSino, k + 0.5f, T + 0.5f);
	  }

	  image[(j)*wdI + (wdI-1-i)] 			= (cs1*dth);
	  image[(j)*wdI + (wdI-1-i-1)] 		= (cs2*dth);
	  image[(j+1)*wdI + (wdI-1-i)]		= (cs3*dth);
	  image[(j+1)*wdI + (wdI-1-i-1)] 		= (cs4*dth);
  }
}
}



extern "C" {
void raft_backprojection_gpu_function(float *d_output, float *d_input, int sizeImage, int nrays, int nangles){


	float dt  = 2.0/(nrays-1);
	float dth = PI/(nangles);
	float delta = (float) 2*fabsf(INIC)/(sizeImage-1);

    // Allocate CUDA array in device memory (sinogram matrix)
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0,hipChannelFormatKindFloat);
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, nangles, nrays);

    // Copy to device memory the sinogram matrix
    hipMemcpyToArray(cuArray, 0, 0, d_input, nrays * nangles * sizeof(float) , hipMemcpyDeviceToDevice);

    // Set texture parameters
    texSino.addressMode[0] = hipAddressModeBorder;
    texSino.addressMode[1] = hipAddressModeBorder;
    texSino.filterMode     = hipFilterModeLinear;
    /*texSino.normalized     = true; */

    // Bind the array to the texture reference
    hipBindTextureToArray(texSino, cuArray, channelDesc);

    //GRID and BLOCKS SIZE
    dim3 threadsPerBlock(TPBXb,TPBYb);
    dim3 grid((sizeImage/threadsPerBlock.x)/2 + 1, (sizeImage/threadsPerBlock.y)/2 + 1);

    //KERNEL EXECUTION
    raft_backprojection_gpu_kernel<<<grid, threadsPerBlock>>>(d_output, sizeImage, nrays, nangles, delta, dt, dth);
    hipDeviceSynchronize();


    hipUnbindTexture(texSino);
    hipFreeArray(cuArray);
}
}
